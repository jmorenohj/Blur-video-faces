#include "hip/hip_runtime.h"

#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>
#include <opencv2/videoio.hpp>
#include <opencv2/highgui.hpp>
#include <iostream>
#include <stdio.h>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace cv;
using namespace std;
#define SHARED_SIZE 125

//*(arr+3*i*1280+3*j+2)
__global__ void blur(uchar *frame,int x, int y, int height, int width, int kernelSize, int totalRows, int totalCols){
    

    __shared__ uchar temp[SHARED_SIZE*SHARED_SIZE*3];
    int initial_row = y-kernelSize/2;
    for(int col = x-kernelSize/2;col<=x+width+kernelSize/2;col++){
        temp[threadIdx.x*SHARED_SIZE*3+(col-(x-kernelSize/2))*3] = frame[(initial_row+threadIdx.x)*3*totalCols+col*3];
        temp[threadIdx.x*SHARED_SIZE*3+(col-(x-kernelSize/2))*3+1] = frame[(initial_row+threadIdx.x)*3*totalCols+col*3+1];
        temp[threadIdx.x*SHARED_SIZE*3+(col-(x-kernelSize/2))*3+2] = frame[(initial_row+threadIdx.x)*3*totalCols+col*3+2];
    }
    __syncthreads();
    uchar acumR = 0, acumG = 0, acumB = 0;
    for(int col = 0;col<=width+kernelSize-1;col++){
        if(col>=kernelSize){
            temp[threadIdx.x*SHARED_SIZE*3 + (col-kernelSize/2)*3] = acumR;
            temp[threadIdx.x*SHARED_SIZE*3 + (col-kernelSize/2)*3+1] = acumG;
            temp[threadIdx.x*SHARED_SIZE*3 + (col-kernelSize/2)*3+2] = acumB;
            acumR -= temp[threadIdx.x*SHARED_SIZE*3 + (col-kernelSize)*3]/kernelSize;
            acumG -= temp[threadIdx.x*SHARED_SIZE*3 + (col-kernelSize)*3+1]/kernelSize;
            acumB -= temp[threadIdx.x*SHARED_SIZE*3 + (col-kernelSize)*3+2]/kernelSize;
        }
        acumR += temp[threadIdx.x*SHARED_SIZE*3+col*3]/kernelSize;
        acumG += temp[threadIdx.x*SHARED_SIZE*3+col*3+1]/kernelSize;
        acumB += temp[threadIdx.x*SHARED_SIZE*3+col*3+2]/kernelSize;
    }
    __syncthreads();
    acumR = 0, acumG = 0, acumB = 0;
    for(int row = 0;row<=height+kernelSize-1;row++){
        if(row>=kernelSize){
            temp[(row-kernelSize/2)*SHARED_SIZE*3 + threadIdx.x*3] = acumR;
            temp[(row-kernelSize/2)*SHARED_SIZE*3 + threadIdx.x*3+1] = acumG;
            temp[(row-kernelSize/2)*SHARED_SIZE*3 + threadIdx.x*3+2] = acumB;
            acumR -= temp[(row-kernelSize)*SHARED_SIZE*3 + threadIdx.x*3]/kernelSize;
            acumG -= temp[(row-kernelSize)*SHARED_SIZE*3 + threadIdx.x*3+1]/kernelSize;
            acumB -= temp[(row-kernelSize)*SHARED_SIZE*3 + threadIdx.x*3+2]/kernelSize;
        }
        acumR += temp[row*SHARED_SIZE*3+threadIdx.x*3]/kernelSize;
        acumG += temp[row*SHARED_SIZE*3+threadIdx.x*3+1]/kernelSize;
        acumB += temp[row*SHARED_SIZE*3+threadIdx.x*3+2]/kernelSize;
    }
    __syncthreads();
    
    
    for(int col = x-kernelSize/2;col<=x+width+kernelSize/2;col++){
        frame[(initial_row+threadIdx.x)*3*totalCols+col*3] = temp[threadIdx.x*SHARED_SIZE*3+(col-(x-kernelSize/2))*3] ;
        frame[(initial_row+threadIdx.x)*3*totalCols+col*3+1] = temp[threadIdx.x*SHARED_SIZE*3+(col-(x-kernelSize/2))*3+1] ;
        frame[(initial_row+threadIdx.x)*3*totalCols+col*3+2] = temp[threadIdx.x*SHARED_SIZE*3+(col-(x-kernelSize/2))*3+2] ;
    }
}

int main(int argc, char *argv[]){
    // Se definen los directorios en donde se lee y se escribe
    char path[100] = "";
    char path2[100] = "";
    strcat(path,argv[1]);
    strcat(path2,argv[2]);

    // Almacena parámetros del video de entrada
    VideoCapture cap(path);
    int frame_width = (int)(cap.get(3));
    int frame_height = (int)(cap.get(4));
    Size frame_size(frame_width, frame_height);
    int fps = 20;
    int totalFrames = (int)cap.get(7);

    // Establece parámetros del video de salida
    VideoWriter output(path2, VideoWriter::fourcc('M', 'P', '4', 'V'),fps, frame_size);

    cout<<"Total frames "<<totalFrames<<endl;
    cout<<"Frame width "<<frame_width<<endl;
    cout<<"Frame height "<<frame_height<<endl;

    // Verifica si se abrió el video con éxito
    if(!cap.isOpened()){
        cout << "Error opening video stream or file" << endl;
        return -1;
    }
    

    // Verifica si se abrió el video con éxito
    int kernelSize = 15;
    double div = (double)1/(kernelSize*kernelSize);


    
    // Variable para almacenar coordenadas y medidas de rostros
    vector<Rect> faces;
    CascadeClassifier face_cascade;
    face_cascade.load("/content/build/blur/haarcascade_frontalface_alt.xml");
    hipError_t err = hipSuccess;
    // Se itera por todos los frames del video
    int cont = 0;
    int blocks_num = 1;

    while(cap.isOpened()){
        // Establece fotograma a analizar
        Mat frame;
        bool isSuccess = cap.read(frame);
        
        // Verifica si el frame se leyó con éxito
        if (!isSuccess){
            cout << "Stream disconnected" << endl;
            break;
        }
            
        if (frame.empty())break;
        // Detecta los rostros en el fotograma
        face_cascade.detectMultiScale(frame, faces, 1.1, 3,0);
        // Itera sobre todos los rostros detectados
        if(faces.empty()){
          output.write(frame);
          continue;
        }

        uchar *h_frame, *d_frame;
        int size,threads_num = 0;
        h_frame = frame.isContinuous()? frame.data: frame.clone().data;
        uint length = frame.total()*frame.channels();        
        size = sizeof(uchar)*length;
        err = hipMalloc((void **)&d_frame, size);
        if (err != hipSuccess){
            fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        err = hipMemcpy(d_frame, h_frame, size, hipMemcpyHostToDevice);
        if (err != hipSuccess){
            fprintf(stderr, "Failed to copy vector h_frame from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        //cout<<faces.size()<<endl;
        for(Rect r:faces){
            if(r.height+kernelSize>=SHARED_SIZE || r.width+kernelSize>=SHARED_SIZE)continue;
            //cout<<r.height<<" "<<r.width<<endl;
            threads_num = r.height+kernelSize-1;
            blur<<<blocks_num,threads_num>>>(d_frame,r.x,r.y,r.height,r.width,kernelSize,frame_height,frame_width);
        }
        hipDeviceSynchronize();
       
        err = hipGetLastError();
        if (err != hipSuccess){
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipMemcpy(h_frame, d_frame, size, hipMemcpyDeviceToHost);
        if (err != hipSuccess){
            fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        

        err = hipFree(d_frame);
        if (err != hipSuccess){
            fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        
    
        
        
        // Limpia el vector de rostros
        faces.clear();
        output.write(frame);

        // Muestra porcentaje de avance
        //cout<<(double)cont*100/totalFrames<<"%"<<endl;
        cont++;
        

        
    }
    
    err = hipDeviceReset();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Finaliza el programa
    
    cap.release();
    destroyAllWindows();
    return 0;
}
